#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_SIZE  16
//#define NUM_STREAMS 100

/* used in streams */
const float *shost_input;
const float *shost_output;

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil(Width_out/(TILE_SIZE*1.0));
    //int H_grid = Height_out / TILE_SIZE;
    int n = blockIdx.x;
    int m = blockIdx.y;
    int h = (blockIdx.z / W_grid) * TILE_SIZE + threadIdx.y;
    int w = (blockIdx.z % W_grid) * TILE_SIZE + threadIdx.x;
    if (h < Height_out && w < Width_out) {
        float acc = 0.0f;
        for (int c = 0; c < Channel; c++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    acc += in_4d(n, c, h+p, w+q) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(n, m, h, w) = acc;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

__host__ void stream_execute(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K);
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    
    /* allocated memory with a double pointer */
    hipMalloc((void**)device_mask_ptr, Channel*Map_out*K*K*sizeof(float));
    hipMalloc((void**)device_input_ptr, Batch*Height*Width*Channel*sizeof(float));
    hipMalloc((void**)device_output_ptr, Batch*(Height-K+1)*(Width-K+1)*Map_out*sizeof(float));

    /* workaround so we have access to the host data */
    shost_input = host_input;
    shost_output = host_output;

    /* copy memory using a single pointer */
    hipMemcpy(*device_mask_ptr, host_mask, Channel*Map_out*K*K*sizeof(float), hipMemcpyHostToDevice);
    hipHostRegister((void*)host_input, Batch*Height*Width*Channel*sizeof(float), hipHostRegisterDefault);
    hipHostRegister((void*)host_output, Batch*(Height-K+1)*(Width-K+1)*Map_out*sizeof(float), hipHostRegisterDefault);

    stream_execute(*device_output_ptr, *device_input_ptr, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
}

/* this is a workaround function so we can do all the memcpying and kernel launching with streams */
__host__ void stream_execute(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K) {
    int W_out = Width - K + 1;
    int H_out = Height - K + 1;
    int W_grid = ceil(W_out/(TILE_SIZE*1.0));
    int H_grid = ceil(H_out/(TILE_SIZE*1.0));
    int Z = H_grid * W_grid;
    int NUM_STREAMS = 10;
    dim3 BlockDim(TILE_SIZE, TILE_SIZE, 1);
    dim3 GridDim(Batch/NUM_STREAMS, Map_out, Z);
    hipStream_t streams[NUM_STREAMS];

    int sinput_size = (Batch*Height*Width*Channel)/NUM_STREAMS;
    int soutput_size = (Batch*(Height-K+1)*(Width-K+1)*Map_out)/NUM_STREAMS;

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        hipMemcpyAsync((void*)(device_input + i*sinput_size), shost_input+(i*sinput_size), sinput_size*sizeof(float), hipMemcpyHostToDevice, streams[i]);
        //hipStreamSynchronize(streams[i]);
        conv_forward_kernel<<<GridDim, BlockDim, 0, streams[i]>>>(device_output+(i*soutput_size), device_input+(i*sinput_size), device_mask, Batch, Map_out, Channel, Height, Width, K);
        //hipStreamSynchronize(streams[i]);
        hipMemcpyAsync((void*)(shost_output+(i*soutput_size)), (device_output + i*soutput_size), soutput_size*sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        //hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
    hipDeviceSynchronize();
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /* And since con_forward_gpu host function does not have the host vectors to copy over to the device, you'll need to shift your host side implementation to a host function which does have those host vectors. - campuswire #512*/
    return;
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    hipHostFree((void*)shost_input);
    hipHostFree((void*)shost_output);
    hipFree(device_mask);
    hipFree(device_input);
    hipFree(device_output);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}