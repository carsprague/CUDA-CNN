#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define TILE_SIZE 8 
//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];
__global__ void conv3d(float *input, float *output, const int z_size, const int y_size, const int x_size) {
  //@@ Insert kernel code here
  __shared__ float subTile[TILE_SIZE+MASK_WIDTH-1][TILE_SIZE+MASK_WIDTH-1][TILE_SIZE+MASK_WIDTH-1];

  // note that threadIdx is relative to the tile, not the entire input
  // e.g. 0 -> TILE_SIZE+(MASK_WIDTH-1)
  int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;

  int radius = MASK_WIDTH / 2;

  int row_o = blockIdx.y * TILE_SIZE + ty;
  int col_o = blockIdx.x * TILE_SIZE + tx;
  int dep_o = blockIdx.z * TILE_SIZE + tz;
  int row_i = row_o - radius;
  int col_i = col_o - radius;
  int dep_i = dep_o - radius;
  // width = x_size, height = y_size, depth = z_size
  if ((row_i >= 0) && (row_i < y_size) && (col_i >= 0) && (col_i < x_size) && (dep_i >= 0) && (dep_i < z_size)) {
    subTile[tz][ty][tx] = input[dep_i*y_size*x_size+row_i*x_size+col_i];
  } else {
    subTile[tz][ty][tx] = 0.0;
  }
  __syncthreads();

  float pvalue = 0;
  if (tx < TILE_SIZE && ty < TILE_SIZE && tz < TILE_SIZE) {
    for (int i = 0; i < MASK_WIDTH; ++i) {
      for (int j = 0; j < MASK_WIDTH; ++j) {
        for (int k = 0; k < MASK_WIDTH; ++k) {
          pvalue += subTile[tz+i][ty+j][tx+k] * M[i][j][k];
        }
      }
    }
    if (row_o < y_size && col_o < x_size && dep_o < z_size) {
      output[dep_o*y_size*x_size+row_o*x_size+col_o] = pvalue;
    }
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  //@@ Initial deviceInput and deviceOutput here.
  float *deviceInput;
  float *deviceOutput;
  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void**)&deviceInput, (inputLength-3) * sizeof(float));
  hipMalloc((void**)&deviceOutput, (inputLength-3) * sizeof(float));

  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, hostInput+3, (inputLength-3)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M), hostKernel, kernelLength * sizeof(float));

  //@@ Initialize grid and block dimensions here
  dim3 DimBlock(TILE_SIZE+(MASK_WIDTH-1),TILE_SIZE+(MASK_WIDTH-1),TILE_SIZE+(MASK_WIDTH-1));
  dim3 DimGrid(ceil(x_size/(1.0*TILE_SIZE)),
               ceil(y_size/(1.0*TILE_SIZE)),
               ceil(z_size/(1.0*TILE_SIZE)));
  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();

  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput+3, deviceOutput, (inputLength-3)*sizeof(float), hipMemcpyDeviceToHost);

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  //@@ Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}

