#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < len) out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  // note that hipMalloc does not return a pointer, but rather returns hipError_t
  // hipMalloc(void** devPtr, size_t size)
  hipError_t err;
  err = hipMalloc((void**)&deviceInput1, inputLength * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed hipMalloc deviceInput1\n");
    exit(EXIT_FAILURE);
  }
  err = hipMalloc((void**)&deviceInput2, inputLength * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed hipMalloc deviceInput2\n");
    exit(EXIT_FAILURE);
  }
  err = hipMalloc((void**)&deviceOutput, inputLength * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed hipMalloc deviceOutput\n");
    exit(EXIT_FAILURE);
  }

  //@@ Copy memory to the GPU here
  // hipMemcpy(void* dest, const void* src, size_t count, hipMemcpyKind kind)
  err = hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed hipMemcpy deviceInput1\n");
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed hipMemcpy deviceInput2\n");
    exit(EXIT_FAILURE);
  }

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(inputLength * sizeof(float)/256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  err = hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed hipMemcpy hostOutput\n");
    exit(EXIT_FAILURE);
  }

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
